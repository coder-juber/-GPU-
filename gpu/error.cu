
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

hipError_t ErrorCheck(hipError_t error_code, const char* filename, int lineNum){
    if(error_code != hipSuccess){
        printf("CUDA Error happen, Error code = %d, name = %s, description = %s\r\nfile = %s, line = %d\r\n",
        error_code, hipGetErrorName(error_code), hipGetErrorString(error_code), filename, lineNum);
        return error_code;
    }
    return error_code;
}

void setGPU(){

    //检测
    int deviceCount = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&deviceCount), __FILE__, __LINE__);

    if(error != hipSuccess || deviceCount == 0){
        printf("no CUDA GPU found\n");
        exit(-1);
    }
    else{
        printf("The count of GPU is %d\n", deviceCount);
    }

    //设置
    int Dev = 0;//默认使用0号GPU
    error = ErrorCheck(hipSetDevice(Dev), __FILE__, __LINE__);
    if(error != hipSuccess){
        printf("set fail\n");
        exit(-1);
    }
    else{
        printf("set success\n");
    }

}

