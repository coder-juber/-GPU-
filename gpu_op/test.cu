#include "error.cuh"

int main(){
    setGPU();
    hipDeviceProp_t prop;
    ErrorCheck(hipGetDeviceProperties(&prop, 0), __FILE__, __LINE__);

    printf("Compute Capability = %d.%d\n", prop.major,prop.minor);
    return 0;
}
